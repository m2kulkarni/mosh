
#include <hip/hip_runtime.h>
#include <stdio.h>


// Kernel function to add the elements of two arrays
__global__ void vecAddKernel(float* A, float* B, float* C, int n)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i<n)  
    {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* h_A, float*h_B, float* h_C, int n)
{
    int size = n*sizeof(float);
    float *d_A, *d_B, *d_C;

    hipError_t err = hipMalloc( (void**) &d_A, size);
    if(err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMalloc( (void**) &d_B, size);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMalloc( (void**) &d_C, size);

    dim3 dimGrid(32,1,1);
    dim3 dimBlock(128,1,1);

    vecAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    for(int i=0; i<n; i++)
        printf("%f ", h_C[i]);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

int main()
{
    int n = 6;
    float h_A[] = {1,2,3,3,2,4};
    float h_B[] = {1,2,3,4,5,3};
    float h_C[] = {0,0,0,0,0,0};    
    vecAdd(h_A, h_B, h_C, n);
    hipDeviceReset();
}

