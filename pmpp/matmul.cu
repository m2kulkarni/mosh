#include "hip/hip_runtime.h"
#include <stdlib.h>

__global__
void matMulKernel(float* out, float* M, float* N, int n)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    for(int j=0; j<=n; j++)
    {
        out[row*n+col] += M[row*n + col]*N[k*n+col];
    }
}

void matMul(float *h_out, float* h_M, float* h_N, int n)
{

    float* d_M, d_N, d_out;
    hipMalloc((void**)&d_M, n*n*sizeof(float));
    hipMemcpy(d_M, h_M, n*n*sizeof(float), hipMemcpyHostToDevice)
    hipMalloc((void**)&d_N, n*n*sizeof(float));
    hipMemcpy(d_N, h_N, n*n*sizeof(float), hipMemcpyHostToDevice)

    hipMalloc((void**)&d_out, n*n*sizeof(float));
    dim3 dimGrid(ceil(n/16.0), ceil(n/16.0));
    dim3 dimBlock(16,16,1);
    matMulKernel<<<dimGrid, dimBlock>>>(d_out, d_M, d_N, n);
    hipMemcpy(h_out, d_out, n*n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_out);
}

int main() 
{
    int n;
    float *h_M, *h_N;
    matMul(h_M, h_N, n);
    hipDeviceReset();
}
