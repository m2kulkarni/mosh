#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define TILE_WIDTH 2

void tiledMatmul(float* h_A, float* h_B, float* h_C, int k, int l, int m);

__global__ void tiledMatmulKernel(float* d_A, float* d_B, float* d_C, int k, int l, int m)
{
    __shared__ float* Mds;
    __shared__ float* Nds;

    Mds = (float*)malloc(sizeof(float)*TILE_WIDTH*TILE_WIDTH);
    Nds = (float*)malloc(sizeof(float)*TILE_WIDTH*TILE_WIDTH);

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by*TILE_WIDTH + ty;  // 0 to k
    int col = bx*TILE_WIDTH + tx; // 0 to m

    float Pval = 0.0f;
    for(int ph=0; ph < ceil(l/TILE_WIDTH); ph++)
    {
        if(row < k && (ph*TILE_WIDTH+tx) < l)
            Mds[ty*TILE_WIDTH + tx] = d_A[row*l + ph*TILE_WIDTH + tx];
        else
            Mds[ty*TILE_WIDTH + tx] = 0.0f;
        
        if(col < m && (ph*TILE_WIDTH+ty) < l)
            Nds[ty*TILE_WIDTH + tx] = d_B[(ph*TILE_WIDTH + ty)*m + col];
        else
            Nds[ty*TILE_WIDTH + tx] = 0.0f;

        __syncthreads();

        for(int i = 0; i < TILE_WIDTH; i++)
        {
            Pval += Mds[ty*TILE_WIDTH + i]*Nds[i*TILE_WIDTH + tx];
        }

    }

    if(row < k && col < m)
        d_C[row*m + col] = Pval;

}

void tiledMatmul(float* h_A, float* h_B, float* h_C, int k, int l, int m)
{
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeof(float)*k*l);
    hipMemcpy(d_A, h_A, sizeof(float)*k*l, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_B, sizeof(float)*l*m);
    hipMemcpy(d_B, h_B, sizeof(float)*l*m, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_C, sizeof(float)*k*l);

    int t = 4;
    dim3 dimGrid(t, t, 1);
    dim3 dimBlock(ceil(m/t), ceil(k/t), 1);
    tiledMatmulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, k, l, m);
    hipMemcpy(h_C, d_C, sizeof(float)*k*m, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void initOnes(float *M, int a, int b)
{
    for(int i=0; i<a; i++)
    {
        for(int j=0; j<b; j++)
        {
            M[i*b + j] = i;
        }
    }

}

void initIdentity(float *M, int a, int b)
{
    for(int i=0; i<a; i++)
    {
        for(int j=0; j<b; j++)
        {
            if(i==j)
                M[i*b + j] = 2;
        }
    }

}

void printMatrix(float* M, int a, int b)
{
    for(int i=0; i<a; i++)
    {
        for(int j=0; j<b; j++)
        {
            printf("%.2f ", M[i*b + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main()
{
    int k = 10;
    int l = 10;
    int m = 10;
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(sizeof(float)*k*l); // A = kxl matrix
    h_B = (float*)malloc(sizeof(float)*l*m); // B = lxm matrix
    h_C = (float*)malloc(sizeof(float)*k*m); // C = kxm matrix
    initIdentity(h_A, k, l);
    printMatrix(h_A, k, l);
    initOnes(h_B, l, m);
    printMatrix(h_B, l, m);

    tiledMatmul(h_A, h_B, h_C, k, l, m);
    printMatrix(h_C, k, m);

    return 0;
}
